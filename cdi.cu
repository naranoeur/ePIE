#include "hip/hip_runtime.h"
#include "cdi.h"
#include <string.h>

#define THRESHOLD 0.000001f
#define BETA 0.5f
#define BLOCK_WIDTH 32
#define HOLDER 64

/**
 * Notes on memory:
 * Registers: very fast, accesible to threads in a (warp?) - fast
 * Local: located in global memory, is slow, allocated for certain set of block communication -slow
 * __shared__ int sharedVar; 
 * Shared: as fast as registers, available to threads within a block - fast
 * __device__ int global_var; - Global variable - slow
 * __constant__ int constant_var; - fast - limited to 64 KB
 * Texture: artifact inherited from graphics feature. Accesible to cuda pipeline, are 2d array
 * caches, somewhat specialized so have their own issues.
 *
 *
 * __syncthreads(); synchronizes threads in a block
 *
 * extern __shared__ allows for dynamic allocation, specify size as an extra 
 * arguement in for example <<<gridDim, blockDim, 10 * sizeof (float)>>>
 *
 * When loading a variable from global memory, it usually loads 32 contiguous bytes in.
 * So you want to access contiguous data rather than randomly located.
 * So using struct of array is more efficient than array of structs.
 *
 * There is cache.
 *
 * Shared memory is split into banks. Only 1 request to a bank per cycle.
 * So if you have multiple threads make requests to the same bank, it will serialize the process.
 * Only matters for threads in a single warp. Looking up just one value in a memory bank in a warp
 * is ok. Looking up several you take a penalty.
 *
 * Patterns of memory access matter
 */


/**
 * Kernels called by overloaded circshift functions
 * Places elements of arr into buffer appropriately displaced
 */
__global__ void gpu_circshiftKernel (float *buffer, float *arr, shortPt shift, shortPt size);
__global__ void gpu_circshiftKernel (float2 *buffer, float2 *arr, shortPt shift, shortPt size);

/**
 * Makes deep copy of cuObject into cuDest
 */
__global__ void gpu_copy (float2* cuDest, float2* cuObject, shortPt size);

/**
 * Kernel for impositing modulus constraint in the algorithm
 */
__global__ void modConstraint (float2* cuObject, float* cuImage, shortPt size);

/**
 * Kernel contains instructions for the domain constrait
 * Current instructions utilize thresholding and non negativity requirement
 */
__global__ void domainConstraint (float2* cuObjOrig, float2* cuObject, shortPt size);

/**
 * Constructor
 */
void Cdi::initCdi (unsigned int width) {

	// ****** Instantiate class variables ****** //
	N = width;
	length = N * N;
	size.x = N;
	size.y = N;
	numIter = 150;
	error = (float*) malloc (numIter * sizeof (float));
	
	// ****** Create the initial guess ****** //
	object = (hipfftComplex*) malloc (length * sizeof (hipfftComplex));
	float x,y;
	int index;
	//float distSq;
	float radius = 0.2 * N;
	//float radSq = radius * radius;

	for (int i = 0; i < N; i++) {
		for (int j =0; j < N; j++) {

			 x = (float)i - (float)N / 2;
			 y = (float)j - (float)N / 2;
			 index = i * N + j;
			 //distSq = x*x + y*y;
			//if (distSq < radSq) {
			if (abs(x) < radius && abs(y) < radius) {
				object[index].x = 1;
			}
			else {
				object[index].x = 0;
			}
			object[index].y = 0;
		}
	}
	
}

// Deconstructor
void Cdi::destroy () {
	free (object);
	free (error);
}

// Runs the algorithm
void Cdi::solve (float* image) {

	// Init gpu variable to hold the image
	float *cuImage;
	errHandle (hipMalloc ((void**) &cuImage, length * sizeof (float)), 19);	
	errHandle (hipMemcpy (cuImage, image, length * sizeof (float), hipMemcpyHostToDevice), 55);

	// Init gpu variables for object estimation using the algorithm
	hipfftComplex *cuObject;
	hipfftComplex *cuHolder;
	errHandle (hipMalloc ((void**) &cuObject, length * sizeof (hipfftComplex)), 21);
	errHandle (hipMalloc ((void**) &cuHolder, length * sizeof (hipfftComplex)), 25);	
	errHandle (hipMemcpy (cuObject, object, sizeof (hipfftComplex) * length, hipMemcpyHostToDevice), 22);
	
	// Create plan used by CUDA's fft function
	hipfftHandle plan;
	if( hipfftPlan2d (&plan, N, N, HIPFFT_C2C) != HIPFFT_SUCCESS){
	    	fprintf (stderr, "CUFFT error: Plan creation failed");		    	
	    	//exit;
	}
	
	// Geometry of the Kernel launch
	//dim3 dimBlock(1);
	//dim3 dimGrid(1);
	if (N > BLOCK_WIDTH) {
		if ((N % BLOCK_WIDTH) == 0) {
			dimBlock.x = BLOCK_WIDTH;
			dimBlock.y = BLOCK_WIDTH;
			dimGrid.x = N / BLOCK_WIDTH;
			dimGrid.y = N / BLOCK_WIDTH;
		} else {
			printf("Error: Invalid image dimensions. Image size has to be a power of 2.\n");
		}
	} else {
		dimBlock.x = size.x;
		dimBlock.y = size.y;
	}

	for (int i = 0; i < numIter; i++) {
		gpu_copy<<<dimGrid, dimBlock>>>(cuHolder, cuObject, size);
		gpu_fft2d (&cuHolder, &plan, size);
		modConstraint<<<dimGrid, dimBlock>>>(cuHolder, cuImage, size);
		gpu_ifft2d (&cuHolder, &plan, size);
		domainConstraint<<<dimGrid, dimBlock>>> (cuObject, cuHolder, size);
	}

	errHandle (hipMemcpy (object, cuObject, sizeof (hipfftComplex) * length, hipMemcpyDeviceToHost), 30);
	//errHandle (hipMemcpy (object, cuHolder, sizeof (hipfftComplex) * length, hipMemcpyDeviceToHost), 30);
	
	hipFree (cuObject);
	hipFree (cuHolder);
	hipFree (cuImage);
	hipfftDestroy (plan);

}

__global__ void gpu_copy (float2* cuDest, float2* cuObject, shortPt size) {

	int tx = blockIdx.x * blockDim.x + threadIdx.x;
	int ty = blockIdx.y * blockDim.y + threadIdx.y;
	// Matrix element index examined by the particular thread running
	int index = tx * blockDim.y * gridDim.y + ty;
	
	cuDest[index] = cuObject[index];
}

// Try to reduce the number of conditional statements as it decreases performance
__global__ void domainConstraint (float2* cuObject, float2* cuHolder, shortPt size) {

	int tx = blockIdx.x * blockDim.x + threadIdx.x;
	int ty = blockIdx.y * blockDim.y + threadIdx.y;
	// Matrix element index examined by the particular thread running
	int index = tx * blockDim.y * gridDim.y + ty;
	
	// Note I normalize the inverse fourier transform here
	float length = size.x * size.y;
	float real = cuHolder[index].x / length;
	float imag = cuHolder[index].y / length;
	
	if (abs(real) < THRESHOLD)
		real = 0;
	if (abs(imag) < THRESHOLD) 
		imag = 0;
	
	if (real < 0) {
		cuObject[index].x = cuObject[index].x - BETA * real;
		cuObject[index].y = cuObject[index].y - BETA * imag;		
	} else {
			
		cuObject[index].x = real;
		cuObject[index].y = imag;
	}
	
	
}

__global__ void modConstraint (float2* cuObject, float* cuImage, shortPt size) {

	int tx = blockIdx.x * blockDim.x + threadIdx.x;
	int ty = blockIdx.y * blockDim.y + threadIdx.y;
	// Matrix element index examined by the particular thread running
	int index = tx * blockDim.y * gridDim.y + ty;
	
	float real = cuObject[index].x;
	float imag = cuObject[index].y;

	float absol = sqrt(real*real + imag*imag);
	if (absol > THRESHOLD) {
		float scale = cuImage[index] / absol;
		cuObject[index].x = scale * real;
		cuObject[index].y = scale * imag;
	} else {
		cuObject[index].x = 0;
		cuObject[index].y = 0;
	}
	
}

//-------------------------------------------------------------------------------------------------------//
//------------------------------- FFT & FFTSHIFT IMPLEMENTATION -----------------------------------------//
//-------------------------------------------------------------------------------------------------------//

void Cdi::gpu_fft2d (float2** cuHolder, hipfftHandle* plan, shortPt size) {
	gpu_ifftshift (cuHolder, size);
	//Fourier Transform
	if (hipfftExecC2C (*plan, *cuHolder, *cuHolder, HIPFFT_FORWARD) != HIPFFT_SUCCESS){
		fprintf (stderr, "CUFFT error: ExecC2C Forward failed");
		//exit;
	}
	gpu_fftshift (cuHolder, size);
}

void Cdi::gpu_ifft2d (float2** cuHolder, hipfftHandle* plan, shortPt size) {
	gpu_ifftshift (cuHolder, size);
	if (hipfftExecC2C (*plan, *cuHolder, *cuHolder, HIPFFT_BACKWARD) != HIPFFT_SUCCESS){
		fprintf (stderr, "CUFFT error: ExecC2C Forward failed");
		//exit;
	}
	gpu_fftshift (cuHolder, size);
}

void Cdi::gpu_fftshift (float **arr, shortPt arrSize) {
	shortPt shift;
	shift.x = arrSize.x / 2;
	shift.y = arrSize.y / 2;
	gpu_circshift (arr, shift, arrSize);
	
}

void Cdi::gpu_ifftshift (float2 **arr, shortPt arrSize) {
	shortPt shift;
	shift.x = (unsigned short) ceil ((float) arrSize.x / 2);
	shift.y = (unsigned short) ceil ((float) arrSize.y / 2);
	gpu_circshift (arr, shift, arrSize);
	
}

void Cdi::gpu_fftshift (float2 **arr, shortPt arrSize) {
	shortPt shift;
	shift.x = arrSize.x / 2;
	shift.y = arrSize.y / 2;
	gpu_circshift (arr, shift, arrSize);
	
}

void Cdi::gpu_ifftshift (float **arr, shortPt arrSize) {
	shortPt shift;
	shift.x = (unsigned short) ceil ((float) arrSize.x / 2);
	shift.y = (unsigned short) ceil ((float) arrSize.y / 2);
	gpu_circshift (arr, shift, arrSize);
	
}

void Cdi::gpu_circshift (float **arr, shortPt shift, shortPt arrSize) {

	float *holder;
	errHandle (hipMalloc ((void**) &holder, sizeof (float) * arrSize.x * arrSize.y), 1);
	gpu_circshiftKernel<<<dimGrid, dimBlock>>>(holder, *arr, shift, arrSize);
	hipFree(*arr);
	*arr = holder;
	
}

void Cdi::gpu_circshift (float2 **arr, shortPt shift, shortPt arrSize) {

	float2 *holder;
	errHandle (hipMalloc ((void**) &holder,  arrSize.x * arrSize.y * sizeof (float2)), 1);
	gpu_circshiftKernel<<<dimGrid, dimBlock>>>( holder, *arr, shift, arrSize);
	hipFree(*arr);
	*arr = holder;
	
}

__global__ void gpu_circshiftKernel (float *buffer, float *arr, shortPt shift, shortPt size) {

	int tx = blockIdx.x * blockDim.x + threadIdx.x;
	int ty = blockIdx.y * blockDim.y + threadIdx.y;
	
	int newX = (tx + shift.x) % size.x;
	int newY = (ty + shift.y) % size.y;
	buffer[newX * size.y + newY] = arr[tx * size.y + ty];
	
}

__global__ void gpu_circshiftKernel (float2 *buffer, float2 *arr, shortPt shift, shortPt size) {

	int tx = blockIdx.x * blockDim.x + threadIdx.x;
	int ty = blockIdx.y * blockDim.y + threadIdx.y;

	int newX = (tx + shift.x) % size.x;
	int newY = (ty + shift.y) % size.y;
	buffer[newX * size.y + newY] = arr[tx * size.y + ty];
	
}

void errHandle (hipError_t err, int i) {
	if (err != hipSuccess){
		printf ("%d: %s \n", i, hipGetErrorString (err));
	}
}
